#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void arrmul(int *x, int *y, int *z)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    z[id] = x[id] * y[id];
}

int main()
{
    int a[3][3];
    int b[3][3];
    int c[3][3];
    int *d, *e, *f;
    int i, j;

    printf("\nEnter nine elements of the first array:\n");
    for (i = 0; i < 3; i++)
    {
        for (j = 0; j < 3; j++)
        {
            scanf("%d", &a[i][j]);
        }
    }

    printf("\nEnter nine elements of the second array:\n");
    for (i = 0; i < 3; i++)
    {
        for (j = 0; j < 3; j++)
        {
            scanf("%d", &b[i][j]);
        }
    }

    hipMalloc((void **)&d, 9 * sizeof(int));
    hipMalloc((void **)&e, 9 * sizeof(int));
    hipMalloc((void **)&f, 9 * sizeof(int));

    hipMemcpy(d, a, 9 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(e, b, 9 * sizeof(int), hipMemcpyHostToDevice);

    arrmul<<<1, 9>>>(d, e, f);

    hipMemcpy(c, f, 9 * sizeof(int), hipMemcpyDeviceToHost);

    printf("\nMultiplication of two arrays:\n");
    for (i = 0; i < 3; i++)
    {
        for (j = 0; j < 3; j++)
        {
            printf("%d\t", c[i][j]);
        }
        printf("\n");
    }

    hipFree(d);
    hipFree(e);
    hipFree(f);

    return 0;
}
